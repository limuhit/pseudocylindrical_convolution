#include "hip/hip_runtime.h"
#include "sphere_uslice.hpp"
#include <hiprand.h>
#include <stdio.h>
#include <math.h>
#include <float.h>
#include "math_functions.hpp"

void sphere_uslice_opt::init(){
    init_base();
}

template <typename scalar_t>
__global__ void init_uslice_param_kernel(const int nthreads, const int npart, const int width, const int * hindex, scalar_t * param){
    CUDA_KERNEL_LOOP(index, nthreads) {
        int ti = index % width;
        int tp = index / width;
        int tw = hindex[tp];
        scalar_t nidx = (ti + 0.5) / width * tw - 0.5 + 1e-9;
        nidx = (nidx<0) ?  nidx + tw : nidx;
        scalar_t nint = static_cast<scalar_t>(static_cast<int>(nidx));
        scalar_t t = nidx - nint;
        scalar_t t2 = t*t;
        scalar_t t3 = t*t2;
        param[index*5] = nint;
        param[index*5+1] = (-t+2*t2-t3)/2; 
        param[index*5+2] = (2-5*t2+3*t3)/2;
        param[index*5+3] = (t+4*t2-3*t3)/2;
        param[index*5+4] = (-t2+t3)/2;   
    }
}

void sphere_uslice_opt::reshape(int num, int channel, int height, int width){
    bool hflag = (height_==height-pad_*2);
    if (!reshape_base(num, channel, height-pad_*2, width-pad_*2)) return; 
    if(hflag) return;
    w_out_ = width_;
    n_out_ = num_ / npart_;
    h_out_ = height_ * npart_;
    hindex_ = at::zeros({npart_}, at::kInt);
    stride_inv_ = at::zeros({npart_}, at::kInt);
    rt_ = 20;
    //printf("%d %d %d \n",h_out_, w_out_, npart_);
    sphere_cal_npart_hw_v3(h_out_, w_out_, npart_, weight_, hindex_.data_ptr<int>());
    int * sw = hindex_.data_ptr<int>();
    int * sidx = stride_inv_.data_ptr<int>();
    for(int i=0;i<npart_;i++){
        sidx[i] = rt_ * width_ / sw[i];
    }
    stride_inv_ = stride_inv_.to(torch::TensorOptions().device(torch::kCUDA, device_));
    hindex_ = hindex_.to(torch::TensorOptions().device(torch::kCUDA, device_));
    //hinv_ = hinv_.to(torch::TensorOptions().device(torch::kCUDA, device_));
    
    init_param_ = true;
    init_inv_ = true;
}

void sphere_uslice_opt::reshape_top(at::TensorOptions option){
    std::vector<std::vector<int64_t>> shapes;
    shapes.push_back({n_out_,channel_, h_out_, w_out_});
    reshape_top_base(option,shapes);
    if(init_param_) resize_param_ = torch::zeros({npart_,width_,5},option);
}

void sphere_uslice_opt::reshape_bottom(at::TensorOptions option){
    std::vector<std::vector<int64_t>> shapes;
    shapes.push_back({num_,channel_,height_+2*pad_,width_+2*pad_});
    reshape_bottom_base(option,shapes);
    if(init_inv_) inv_param_ = at::zeros({npart_,width_*rt_}, option);
}


template <typename scalar_t>
__global__ void sphere_uslice_forward_kernel(const int nthreads, const scalar_t* const input,  
    scalar_t * const output, const scalar_t * param, const int * hindex, const int width, 
    const int height, const int height_out, const int channel, const int npart, 
    const int pad, const int stride_h, const int stride_w) {
   CUDA_KERNEL_LOOP(index, nthreads) {
       int tw = index % width;
       int th = (index / width) % height_out;
       int tc = (index / width / height_out) % channel;
       int tn = index / width / height_out / channel;
       int ph = th % height;
       int pb = th / height;
       int pn = tn * npart + pb;
       int base = (pb*width+tw)*5;
       int pw = static_cast<int>(param[base]);
       int pidx = ((pn*channel + tc)*stride_h + ph + pad) * stride_w + pad;
       int wl = hindex[pb];
       if(pw>0 && pw < wl-2){
           output[index] = param[base+1]*input[pidx+pw-1] + param[base+2]*input[pidx+pw] +
                           param[base+3]*input[pidx+pw+1] + param[base+4]*input[pidx+pw+2];
            //printf("%d %d %d %d %d\n", pb, pw-1, pw, pw+1,pw+2);
       }else{
           output[index] = param[base+1]*input[pidx+(pw-1+wl)%wl] + param[base+2]*input[pidx+pw] +
                           param[base+3]*input[pidx+(pw+1)%wl] + param[base+4]*input[pidx+(pw+2)%wl];
            //printf("%d %d %d %d %d\n", pb, (pw-1+wl)%wl, pw, (pw+1)%wl,(pw+2)%wl);
       }
   }
}

std::vector<at::Tensor>  sphere_uslice_opt::forward_cuda(at::Tensor  bottom_data) 
{
    reshape(bottom_data.size(0), bottom_data.size(1), bottom_data.size(2), bottom_data.size(3));
    reshape_top(bottom_data.options());
	int count;
	AT_DISPATCH_FLOATING_TYPES(
		bottom_data.scalar_type(), "sphere_uslice_forward_cuda", 
			([&] {
                    count = width_ * npart_;
                    if(init_param_){
                        init_uslice_param_kernel<< <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, stream_ >> >
                            (count, npart_, width_, hindex_.data_ptr<int>(), resize_param_.data_ptr<scalar_t>());
                        init_param_ = false;
                    }
                    count = n_out_ * channel_ * w_out_ * h_out_;
                    sphere_uslice_forward_kernel<< <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, stream_ >> >
                    (count, bottom_data.data_ptr<scalar_t>(), top_data_[0].data_ptr<scalar_t>(), resize_param_.data_ptr<scalar_t>(), 
                        hindex_.data_ptr<int>(), width_, height_, h_out_, channel_, npart_, pad_, height_+2*pad_, width_+2*pad_);
                    CUDA_POST_KERNEL_CHECK;

   			    }
			)
    );
    return top_data_;
}

template <typename scalar_t>
__global__ void sphere_uslice_init_inv_kernel(const int nthreads, 
    scalar_t * const inv_param, const scalar_t * param, const int * hindex, const int width, 
    const int npart, const int * stride_inv, const int stride) {
   CUDA_KERNEL_LOOP(index, nthreads) {
       int tw = index % width;
       int pb = (index / width) % npart;
       int base = (pb*width+tw)*5;
       int pw = static_cast<int>(param[base]);
       int wl = hindex[pb];
       scalar_t nidx;
       if(pw>0 && pw < wl-2){
           for (int j = -1; j<3; j++){
                nidx = atomicAdd(inv_param+pb*stride+stride_inv[pb]*(pw+j),1.);
                if(nidx*2+2>=stride_inv[pb]) printf("inv_param stack overflow! %d %d %d %d %d\n", pb, stride_inv[pb],tw,pw,j);
                inv_param[pb*stride+stride_inv[pb]*(pw+j)+static_cast<int>(nidx)*2+1] = tw;
                inv_param[pb*stride+stride_inv[pb]*(pw+j)+static_cast<int>(nidx)*2+2] = param[base+j+2];
           }
       }else{
            for (int j = -1; j<3; j++){
                nidx = atomicAdd(inv_param+pb*stride+stride_inv[pb]*((pw+j+wl)%wl),1.);
                if(nidx*2+2>=stride_inv[pb]) printf("inv_param stack overflow! %d %d %d %d %d\n", pb, stride_inv[pb],tw,pw,j);
                inv_param[pb*stride+stride_inv[pb]*((pw+j+wl)%wl)+static_cast<int>(nidx)*2+1] = tw;
                inv_param[pb*stride+stride_inv[pb]*((pw+j+wl)%wl)+static_cast<int>(nidx)*2+2] = param[base+j+2];
            }
       }
   }
}

template <typename scalar_t>
__global__ void sphere_uslice_backward_kernel(const int nthreads, scalar_t* const input,  
    const scalar_t * const output, const int * hindex, const int height, 
    const int height_out, const int channel, const int width, const int npart, 
    const int * stride_inv, const int stride, const  scalar_t * const inv_param, 
    const int pad, const int stride_h, const int stride_w) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        int tw = index % width;
        int th = (index / width) % height;
        int tc = (index / width / height) % channel;
        int tn = index / width / height / channel;
        int tp = tn % npart;
        int pidx = ((tn * channel + tc) * stride_h + th + pad)*stride_w + tw + pad;
        //input[pidx] = 0;
        if(tw>=hindex[tp]) continue;
        int pn = tn / npart;
        int base = ((pn*channel + tc)*height_out + tp*height + th)*width;
        int wbase = tp * stride + tw*stride_inv[tp];
        int num = static_cast<int>(inv_param[wbase]+1e-6);
        for(int i=0;i<num;i++){
            input[pidx] = input[pidx] + output[base+static_cast<int>(inv_param[wbase+i*2+1])]*inv_param[wbase+i*2+2];
        }
    }    
}

std::vector<at::Tensor>  sphere_uslice_opt::backward_cuda(at::Tensor  top_diff) 
{
    reshape_bottom(top_diff.options());
	int count;
	AT_DISPATCH_FLOATING_TYPES(
		top_diff.scalar_type(), "sphere_uslice_backward_cuda", 
			([&] {
                    if(init_inv_){
                        count = npart_ * width_;
                        caffe_gpu_set(stream_, count*rt_, 0, inv_param_.data_ptr<scalar_t>());
                        sphere_uslice_init_inv_kernel<< <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, stream_ >> >
                            (count, inv_param_.data_ptr<scalar_t>(), resize_param_.data_ptr<scalar_t>(), hindex_.data_ptr<int>(), width_, 
                               npart_, stride_inv_.data_ptr<int>(), width_*rt_);
                        init_inv_ = false;
                    }
                    count = num_*channel_*height_*width_;
                    caffe_gpu_set(stream_, num_*channel_*(height_+2*pad_)*(width_+2*pad_),0, bottom_diff_[0].data_ptr<scalar_t>());
                    sphere_uslice_backward_kernel<< <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, stream_ >> >
                    (count, bottom_diff_[0].data_ptr<scalar_t>(), top_diff.data_ptr<scalar_t>(), hindex_.data_ptr<int>(), height_, 
                        h_out_, channel_, width_, npart_,   stride_inv_.data_ptr<int>(), width_*rt_, inv_param_.data_ptr<scalar_t>(), 
                        pad_, height_+2*pad_, width_+2*pad_);
   			    }
			)
    );
    return {bottom_diff_[0],inv_param_,stride_inv_};
}